#include "hip/hip_runtime.h"

#include <stdlib.h>
#include <stdio.h>


#include "power_iteration.cuh"

#define NUM_THREAD 256
#define NUM_BLOCK 4069

template<class T>
struct SharedMemory
{
    __device__ inline operator       T *()
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }

    __device__ inline operator const T *() const
    {
        extern __shared__ int __smem[];
        return (T *)__smem;
    }
};

__global__ void expand_vector(float *d_NNZ_values, float *d_vec, unsigned *d_indices, float* d_expanded_vec,int NNZ, int dim){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    __shared__ float d_expanded_vec_shared[256];
    __shared__ float d_NNZ_values_shared[256];

    while(i < NNZ){
        d_expanded_vec_shared[i%256] = d_vec[d_indices[i]];
        d_NNZ_values_shared[i%256] = d_NNZ_values[i];
        i += blockDim.x*gridDim.x;
    }
    __syncthreads();
    i = blockDim.x * blockIdx.x + threadIdx.x;
    while(i < NNZ){
        d_expanded_vec_shared[i%256] = d_NNZ_values_shared[i%256]*d_expanded_vec_shared[i%256];
        i += blockDim.x*gridDim.x;
    }
    __syncthreads();
    i = blockDim.x * blockIdx.x + threadIdx.x;
    while(i < NNZ){
        d_expanded_vec[i] = d_expanded_vec_shared[i%256];
        i += blockDim.x*gridDim.x;
    }
}

__global__ void extract_vector(float *d_expanded_vec, float *d_vec, unsigned *d_rindices, int dim){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    while(i < dim){
        d_vec[i] = d_expanded_vec[d_rindices[i]];
        i += blockDim.x*gridDim.x;
    }
}

/*
  Modification of the reduce6 code provided in the Cuda examples
  Computes the norm, and normalizes the vector
  g_idata is the input vector
  g_odata stores the value of the norm
  n = size of row
*/

template <class T, unsigned int blockSize, bool nIsPow2>
__global__ void Normalize_Vector(T *g_idata, T *g_odata, unsigned int n){
    T *sdata = SharedMemory<T>();

    // perform first level of reduction,
    // reading from global memory, writing to shared memory
    unsigned int tid = threadIdx.x;
    unsigned int i = blockIdx.x*blockSize*2 + threadIdx.x;
    unsigned int gridSize = blockSize*2*gridDim.x;

    T mySum = 0;

    // we reduce multiple elements per thread.  The number is determined by the
    // number of active thread blocks (via gridDim).  More blocks will result
    // in a larger gridSize and therefore fewer elements per thread
    while (i < n)
    {
        mySum += g_idata[i]*g_idata[i];

        // ensure we don't read out of bounds -- this is optimized away for powerOf2 sized arrays
        if (nIsPow2 || i + blockSize < n)
            mySum += g_idata[i+blockSize]*g_idata[i+blockSize];

        i += gridSize;
    }

    // each thread puts its local sum into shared memory
    sdata[tid] = mySum;
    __syncthreads();


    // do reduction in shared mem
    if ((blockSize >= 512) && (tid < 256))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 256];
    }

    __syncthreads();

    if ((blockSize >= 256) &&(tid < 128))
    {
            sdata[tid] = mySum = mySum + sdata[tid + 128];
    }

     __syncthreads();

    if ((blockSize >= 128) && (tid <  64))
    {
       sdata[tid] = mySum = mySum + sdata[tid +  64];
    }

    __syncthreads();

#if (__CUDA_ARCH__ >= 300 )
    if ( tid < 32 )
    {
        // Fetch final intermediate sum from 2nd warp
        if (blockSize >=  64) mySum += sdata[tid + 32];
        // Reduce final warp using shuffle
        for (int offset = warpSize/2; offset > 0; offset /= 2) 
        {
            mySum += __shfl_down(mySum, offset);
        }
    }
#else
    // fully unroll reduction within a single warp
    if ((blockSize >=  64) && (tid < 32))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 32];
    }

    __syncthreads();

    if ((blockSize >=  32) && (tid < 16))
    {
        sdata[tid] = mySum = mySum + sdata[tid + 16];
    }

    __syncthreads();

    if ((blockSize >=  16) && (tid <  8))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  8];
    }

    __syncthreads();

    if ((blockSize >=   8) && (tid <  4))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  4];
    }

    __syncthreads();

    if ((blockSize >=   4) && (tid <  2))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  2];
    }

    __syncthreads();

    if ((blockSize >=   2) && ( tid <  1))
    {
        sdata[tid] = mySum = mySum + sdata[tid +  1];
    }

    __syncthreads();
#endif

    // write result for this block to global mem
    if (tid == 0) g_odata[blockIdx.x] = sqrt(mySum);
    // Normalize the input vector
    __syncthreads();
    tid = blockDim.x * blockIdx.x + threadIdx.x;
    float temp_val = g_odata[0];
    while(tid < n){
        g_idata[tid] /= temp_val;
        tid += blockDim.x*gridDim.x;
    }
} 

void matvec(const CUDPPHandle scanplan, float *d_NNZ_values, float *d_vec, unsigned *d_indices, unsigned *d_rindices, unsigned *d_flags, float* d_scanned_vec, float* d_expanded_vec,int NNZ, int dim){
    
    int threadsPerBlock = NUM_THREAD;
    int blocksPerGrid = NUM_BLOCK;

    expand_vector<<<blocksPerGrid, threadsPerBlock>>>(d_NNZ_values, d_vec, d_indices, d_expanded_vec, NNZ, dim);
  
    
    cudppSegmentedScan(scanplan,d_scanned_vec,d_expanded_vec,d_flags,NNZ);
    
    extract_vector<<<blocksPerGrid, threadsPerBlock>>>(d_scanned_vec, d_vec, d_rindices, dim);
}

void eigenvalue_solver(csr_mat *h_matrix, float *h_vec){
 
    hipEvent_t start, stop;
    float elapsedTime;
    int NNZ = h_matrix->nnz;
    int dim = h_matrix->rows;
    float *h_expanded_vec = (float *)malloc(sizeof(unsigned)*NNZ);
    float *h_scanned_vec = (float *)malloc(sizeof(unsigned)*NNZ);
    float *d_NNZ_values,*d_vec, *d_expanded_vec,*d_scanned_vec, *d_norm;
    unsigned *d_indices,*d_flags,*d_rindices;
    
    /*Copy stuff from host matrix to device */
    hipMalloc((void **)&d_NNZ_values, sizeof(float)*NNZ);
    hipMalloc((void **)&d_indices, sizeof(unsigned)*NNZ);
    hipMalloc((void **)&d_rindices, sizeof(unsigned)*(dim+1));
    hipMalloc((void **)&d_flags, sizeof(unsigned)*NNZ);
    hipMemcpy(d_NNZ_values, h_matrix->vals, sizeof(float)*NNZ, hipMemcpyHostToDevice);
    hipMemcpy(d_indices, h_matrix->cols, sizeof(unsigned)*NNZ, hipMemcpyHostToDevice);
    hipMemcpy(d_rindices, h_matrix->ptr, sizeof(unsigned)*(dim+1), hipMemcpyHostToDevice);
    hipMemcpy(d_flags, h_matrix->flags, sizeof(unsigned)*NNZ, hipMemcpyHostToDevice);

    /* Stuff for matvec operation */
    hipMalloc((void **)&d_vec, sizeof(float)*dim);
    hipMalloc((void **)&d_expanded_vec, sizeof(float)*NNZ);
    hipMalloc((void **)&d_scanned_vec, sizeof(float)*NNZ);
    hipMalloc((void **)&d_norm, sizeof(float));

    hipMemcpy(d_vec, h_vec, sizeof(unsigned)*dim, hipMemcpyHostToDevice);
    // Launch the Vector Add CUDA Kernel

    printf("--------Initial Vector----------\n");  
    for (int i = 0; i < dim; ++i){
        printf("%f\n",h_vec[i]);
    }
    printf("---------------------\n");  
    CUDPPHandle theCudpp;
    cudppCreate(&theCudpp);

    CUDPPConfiguration config;
    config.op = CUDPP_ADD;
    config.datatype = CUDPP_FLOAT;
    config.algorithm = CUDPP_SEGMENTED_SCAN;
    config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE;

    CUDPPHandle scanplan = 0;
    CUDPPResult res = cudppPlan(theCudpp, &scanplan, config, NNZ, 1, 0);

    float *h_temp_vec = (float *)malloc(sizeof(unsigned)*dim);
    int threads = 256;
    int blocks = 4096;
    dim3 dimBlock(threads, 1, 1);
    dim3 dimGrid(blocks, 1, 1);
    int smemSize = (threads <= 32) ? 2 * threads * sizeof(float) : threads * sizeof(float);

    hipEventCreate(&start);
    hipEventRecord(start,0);

    for(int count = 0; count < 3; count++){
        matvec(scanplan,d_NNZ_values, d_vec, d_indices, d_rindices, d_flags, d_scanned_vec,d_expanded_vec,NNZ,dim);      
        Normalize_Vector<float, NUM_THREAD, false><<< dimGrid, dimBlock, smemSize >>>(d_vec, d_norm, dim);
    
        
        
        /* Used to see vector after each iteration
        hipMemcpy(h_vec, d_vec, dim*sizeof(float), hipMemcpyDeviceToHost);
        printf("Vector after normalizing---------------------\n");  
        for (int i = 0; i < dim; ++i){
            printf("%f\n",h_vec[i]);
        }
        printf("---------------------\n");
        */  
    }
    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start,stop);
    printf("Elapsed time : %f ms\n" ,elapsedTime);
    printf("Looks to be %f GFlops\n",((3*NNZ)*20*0.000001)/(elapsedTime));
    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    hipMemcpy(h_scanned_vec, d_scanned_vec, NNZ*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_expanded_vec, d_expanded_vec, NNZ*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_vec, d_vec, dim*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(d_vec, h_vec, sizeof(unsigned)*dim, hipMemcpyHostToDevice);

    /*
    printf("h_vec after matvec\n");
    for (int i = 0; i < dim; ++i){
        printf("%f\n",h_vec[i]);
    }
    */

/*
    hipMemcpy(h_vec, d_vec, dim*sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < dim; ++i){
        printf("%d --- %f --- %f\n",h_matrix->ptr[i],h_vec[i],h_temp_vec[i]);
    }
    printf("---------------------\n");
*/ 
    /*
    for (int i = 0; i < NNZ; ++i){
        printf("%d %f - %f :: %f --- %d\n",h_matrix->cols[i],h_matrix->vals[i],h_expanded_vec[i],h_scanned_vec[i], h_matrix->flags[i]);
    }
    */
    res = cudppDestroyPlan(scanplan);
    cudppDestroy(theCudpp);
    // Free device global memory
    hipFree(d_NNZ_values);
    hipFree(d_indices);
    hipFree(d_rindices);
    hipFree(d_vec);
    hipFree(d_flags);
    hipFree(d_norm);
    hipFree(d_expanded_vec);
    hipFree(d_scanned_vec);  

    free(h_expanded_vec);
    free(h_temp_vec);
    free(h_scanned_vec);  
}