#include "hip/hip_runtime.h"
/**
 * Copyright 1993-2015 NVIDIA Corporation.  All rights reserved.
 *
 * Please refer to the NVIDIA end user license agreement (EULA) associated
 * with this source code for terms and conditions that govern your use of
 * this software. Any use, reproduction, disclosure, or distribution of
 * this software and related documentation outside the terms of the EULA
 * is strictly prohibited.
 *
 */

/**
 * Vector addition: C = A + B.
 *
 * This sample is a very basic sample that implements element by element
 * vector addition. It is the same as the sample illustrating Chapter 2
 * of the programming guide with some additions like error checking.
 */

#include <stdio.h>

// For the CUDA runtime routines (prefixed with "cuda_")
#include <hip/hip_runtime.h>
#include <sys/time.h>

#include <cudpp.h>

//#include <hip/hip_runtime_api.h>
/**
 * CUDA Kernel Device code
 *
 * Computes the vector addition of A and B into C. The 3 vectors have the same
 * number of elements numElements.
 */
__global__ void expand_vector(float *d_NNZ_values, float *d_vec, unsigned *d_indices, float* d_expanded_vec,int NNZ, int dim){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    while(i < NNZ){
        d_expanded_vec[i] = d_vec[d_indices[i]];
        i += blockDim.x*gridDim.x;
    }
    __syncthreads();
    i = blockDim.x * blockIdx.x + threadIdx.x;
    while(i < NNZ){
        d_expanded_vec[i] = d_NNZ_values[i]*d_expanded_vec[i];
        i += blockDim.x*gridDim.x;
    }

}

__global__ void extract_vector(float *d_expanded_vec, float *d_vec, unsigned *d_rindices, int dim){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    while(i < dim){
        d_vec[i] = d_expanded_vec[d_rindices[i]-1];
        i += blockDim.x*gridDim.x;
    }
}
__global__ void vector_mul(float *d_NNZ_values, float *d_expanded_vec, float* d_vec,int NNZ, int dim){
    int i = blockDim.x * blockIdx.x + threadIdx.x;

    while(i < NNZ){
        d_expanded_vec[i] = d_NNZ_values[i]*d_expanded_vec[i];
        i += blockDim.x*gridDim.x;
    }
}

void matvec(const CUDPPHandle scanplan, float *d_NNZ_values, float *d_vec, unsigned *d_indices, unsigned *d_rindices, unsigned *d_flags, float* d_scanned_vec, float* d_expanded_vec,int NNZ, int dim){
    
    int threadsPerBlock = 256;
    //int blocksPerGrid =(NNZ + threadsPerBlock - 1) / threadsPerBlock;
    int blocksPerGrid = 4096;
    //printf("CUDA kernel launch with %d blocks of %d threads\n", blocksPerGrid, threadsPerBlock);
    
    expand_vector<<<blocksPerGrid, threadsPerBlock>>>(d_NNZ_values, d_vec, d_indices, d_expanded_vec, NNZ, dim);
    
    //vector_mul<<<blocksPerGrid, threadsPerBlock>>>(d_NNZ_values, d_expanded_vec, d_vec, NNZ, dim);
    
    cudppSegmentedScan(scanplan,d_scanned_vec,d_expanded_vec,d_flags,NNZ);
    
    extract_vector<<<blocksPerGrid, threadsPerBlock>>>(d_scanned_vec, d_vec, d_rindices, dim);
}


/**
 * Host main routine
 */
int
main(int argc, char* argv[])
{
    int dim = atoi(argv[1]),NNZ = dim;
    // Allocate the host Arrays
    float *h_NNZ_values = (float *)malloc(sizeof(unsigned)*NNZ);
    unsigned *h_indices = (unsigned *)malloc(sizeof(unsigned)*NNZ);
    unsigned *h_rindices = (unsigned *)malloc(sizeof(unsigned)*dim);
    unsigned *h_flags = (unsigned *)malloc(sizeof(unsigned)*NNZ);
    float *h_vec = (float *)malloc(sizeof(unsigned)*dim);
    float *h_expanded_vec = (float *)malloc(sizeof(unsigned)*NNZ);
    float *h_scanned_vec = (float *)malloc(sizeof(unsigned)*NNZ);

    hipEvent_t start, stop;
    float elapsedTime;
    // Allocate the host output vector C

    // Initialize the host input vectors
    for (int i = 0; i < dim; ++i){
        h_vec[i] = 1.0;
        //printf("%f\n",h_vec[i]);
        h_rindices[i] = (i+1);
    }
    for (int i = 0; i < dim; ++i){
        h_NNZ_values[i] = 1.0;
        h_indices[i] = i;
        h_flags[i] = 1;
    }
    h_NNZ_values[0] = 4.0;
    printf("Done setting up Host arrays\n");
    // Allocate the device arrays
    float *d_NNZ_values,*d_vec, *d_expanded_vec,*d_scanned_vec, *d_norm;
    unsigned *d_indices,*d_flags,*d_rindices;
    
    hipMalloc((void **)&d_NNZ_values, sizeof(float)*NNZ);
    hipMalloc((void **)&d_vec, sizeof(float)*dim);
    hipMalloc((void **)&d_indices, sizeof(unsigned)*NNZ);
    hipMalloc((void **)&d_rindices, sizeof(unsigned)*dim);
    hipMalloc((void **)&d_flags, sizeof(unsigned)*NNZ);
    hipMalloc((void **)&d_expanded_vec, sizeof(float)*NNZ);
    hipMalloc((void **)&d_scanned_vec, sizeof(float)*NNZ);
    hipMalloc((void **)&d_norm, sizeof(float));
    // Copy the host input vectors A and B in host memory to the device input vectors in
    // device memory
    hipMemcpy(d_NNZ_values, h_NNZ_values, sizeof(float)*NNZ, hipMemcpyHostToDevice);
    hipMemcpy(d_vec, h_vec, sizeof(unsigned)*dim, hipMemcpyHostToDevice);
    hipMemcpy(d_flags, h_flags, sizeof(unsigned)*NNZ, hipMemcpyHostToDevice);
    hipMemcpy(d_indices, h_indices, sizeof(unsigned)*NNZ, hipMemcpyHostToDevice);
    hipMemcpy(d_rindices, h_rindices, sizeof(unsigned)*dim, hipMemcpyHostToDevice);

    // Launch the Vector Add CUDA Kernel

    CUDPPHandle theCudpp;
    cudppCreate(&theCudpp);

    CUDPPConfiguration config;
    config.op = CUDPP_ADD;
    config.datatype = CUDPP_FLOAT;
    config.algorithm = CUDPP_SEGMENTED_SCAN;
    config.options = CUDPP_OPTION_FORWARD | CUDPP_OPTION_INCLUSIVE;

    CUDPPHandle scanplan = 0;
    CUDPPResult res = cudppPlan(theCudpp, &scanplan, config, NNZ, 1, 0);
    hipEventCreate(&start);
    hipEventRecord(start,0);

    matvec(scanplan,d_NNZ_values, d_vec, d_indices, d_rindices, d_flags, d_scanned_vec,d_expanded_vec,NNZ,dim);

    hipEventCreate(&stop);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start,stop);
    printf("Elapsed time : %f ms\n" ,elapsedTime);
    printf("Seems to be %f GFlops\n",((3*NNZ)*3*0.000000001)/(elapsedTime*0.001));

    // Copy the device result vector in device memory to the host result vector
    // in host memory.
    hipMemcpy(h_scanned_vec, d_scanned_vec, NNZ*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_expanded_vec, d_expanded_vec, NNZ*sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_vec, d_vec, dim*sizeof(float), hipMemcpyDeviceToHost);
    /*
    for (int i = 0; i < dim; ++i){
        printf("%d --- %f\n",h_rindices[i],h_vec[i]);
    }
    printf("---------------------\n");
    for (int i = 0; i < NNZ; ++i){
        //printf("%d %f - %f :: %f --- %d\n",h_indices[i],h_NNZ_values[i],h_expanded_vec[i],h_scanned_vec[i], h_flags[i]);
    }
    */
    res = cudppDestroyPlan(scanplan);
    cudppDestroy(theCudpp);
    // Free device global memory
    hipFree(d_NNZ_values);
    hipFree(d_indices);
    hipFree(d_rindices);
    hipFree(d_vec);
    hipFree(d_flags);
    hipFree(d_norm);
    hipFree(d_expanded_vec);
    hipFree(d_scanned_vec);

    // Free host memory
    free(h_NNZ_values);
    free(h_indices);
    free(h_rindices);
    free(h_vec);
    free(h_flags);
    free(h_expanded_vec);
    free(h_scanned_vec);

    return 0;
}

